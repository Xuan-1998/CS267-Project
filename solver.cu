#include <iostream>
#include <algorithm>
#include <chrono>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define IX(i, j) ((i) + (N + 2) * (j))
#define SWAP(x0, x)      \
    {                    \
        float *tmp = x0; \
        x0 = x;          \
        x = tmp;         \
    }
#define NUM_THREADS 256

void add_source(int N, float *x, float *s, float dt)
{
    int i, size = (N + 2) * (N + 2);
    for (i = 0; i < size; i++)
        x[i] += dt * s[i];
}

void set_bnd(int N, int b, float *x)
{
    int i;
    for (i = 1; i <= N; i++)
    {
        x[IX(0, i)] = b == 1 ? -x[IX(1, i)] : x[IX(1, i)];
        x[IX(N + 1, i)] = b == 1 ? -x[IX(N, i)] : x[IX(N, i)];
        x[IX(i, 0)] = b == 2 ? -x[IX(i, 1)] : x[IX(i, 1)];
        x[IX(i, N + 1)] = b == 2 ? -x[IX(i, N)] : x[IX(i, N)];
    }
    x[IX(0, 0)] = 0.5 * (x[IX(1, 0)] + x[IX(0, 1)]);
    x[IX(0, N + 1)] = 0.5 * (x[IX(1, N + 1)] + x[IX(0, N)]);
    x[IX(N + 1, 0)] = 0.5 * (x[IX(N, 0)] + x[IX(N + 1, 1)]);
    x[IX(N + 1, N + 1)] = 0.5 * (x[IX(N, N + 1)] + x[IX(N + 1, N)]);
}


__global__ void projectHelper1(int N, float *u, float *v, float *p, float *div) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int i = tid % N + 1, j = tid / N + 1;
    
    // printf();
    float h;
    h = 1.0 / N;
    
    if (i <= N && j <= N) {
        // printf("rows is: %d, cols is %d\n", i, j);
        div[IX(i, j)] = -0.5 * h * (u[IX(i + 1, j)] - u[IX(i - 1, j)] + v[IX(i, j + 1)] - v[IX(i, j - 1)]);
        p[IX(i, j)] = 0;
    } else {
        printf("%d\n", tid);
    }
    __syncthreads();
}
void project(int N, float *u, float *v, float *p, float *div, float *p_new)
{
    int i, j, k;
    float h;
    h = 1.0 / N;

    const int size = (N + 2) * (N + 2);
    int mem_size = size*sizeof(float);

    float *d_u = nullptr;
    float *d_v = nullptr;
    float *d_p = nullptr;
    float *d_div = nullptr;

    hipMalloc(&d_u, sizeof(u));
    hipMemcpy(d_u, u, sizeof(u), hipMemcpyHostToDevice);
    hipMalloc(&d_v, sizeof(v));
    hipMemcpy(d_v, v, sizeof(v), hipMemcpyHostToDevice);
    hipMalloc(&d_p, sizeof(p));
    hipMemcpy(d_p, p, sizeof(p), hipMemcpyHostToDevice);
    hipMalloc(&d_div, sizeof(div));
    hipMemcpy(d_div, div, sizeof(div), hipMemcpyHostToDevice);
    // dim3 gridDim(N / 16, N / 16);
    // dim3 blockDim(16, 16);
    std::cout << "It comes before projectHelper1<<<(size + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(N, u, v, p, div);\n";
    projectHelper1<<<(size + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(N, u, v, p, div);
    std::cout << "It comes after projectHelper1<<<(size + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(N, u, v, p, div);\n";
    // cudaDeviceSynchronize();
    // std::cout << "It comes after cudaDeviceSynchronize";
    hipMemcpy(u, d_u, sizeof(u), hipMemcpyDeviceToHost);
    hipMemcpy(v, d_v, sizeof(v), hipMemcpyDeviceToHost);
    hipMemcpy(p, d_p, sizeof(d_p), hipMemcpyDeviceToHost);
    hipMemcpy(div, d_div, sizeof(div), hipMemcpyDeviceToHost);
    std::cout << "It comes all the way down of copy\n";
    // cudaFree(d_u);
    // cudaFree(d_v);
    // cudaFree(d_p);
    // cudaFree(d_div);
    // let's don't free
    std::cout << "It comes all the way down of free\n";

    // for (i = 1; i <= N; i++)
    // {
    //     for (j = 1; j <= N; j++)
    //     {
    //         div[IX(i, j)] = -0.5 * h * (u[IX(i + 1, j)] - u[IX(i - 1, j)] + v[IX(i, j + 1)] - v[IX(i, j - 1)]);
    //         p[IX(i, j)] = 0;
    //     }
    // }
    
    

    set_bnd(N, 0, div);
    set_bnd(N, 0, p);
    // std::cout << "It comes all the way down of free";

    std::cout << "It goes after bnd setting";

    for (k = 0; k < 20; k++)
    {

        for (i = 1; i <= N; i++)
        {
            for (j = 1; j <= N; j++)
            {
                p_new[IX(i, j)] = (div[IX(i, j)] + p[IX(i - 1, j)] + p[IX(i + 1, j)] +
                                   p[IX(i, j - 1)] + p[IX(i, j + 1)]) /
                                  4;
            }
        }
        SWAP(p, p_new);
        set_bnd(N, 0, p);
    }
    for (i = 1; i <= N; i++)
    {
        for (j = 1; j <= N; j++)
        {
            u[IX(i, j)] -= 0.5 * (p[IX(i + 1, j)] - p[IX(i - 1, j)]) / h;
            v[IX(i, j)] -= 0.5 * (p[IX(i, j + 1)] - p[IX(i, j - 1)]) / h;
        }
    }
    set_bnd(N, 1, u);
    set_bnd(N, 2, v);
}

void diffuse(int N, int b, float *x, float *x0, float diff, float dt)
{
    int i, j, k;
    float a = dt * diff * N * N;
    // std::cout << "a:" << a << ", dt:" << dt << ", diff:" << diff << std::endl;
    for (i = 1; i <= N; i++)
    {
        for (j = 1; j <= N; j++)
        {
            x[IX(i, j)] = x0[IX(i, j)] + a * (x0[IX(i + 1, j)] + x0[IX(i - 1, j)] + x0[IX(i, j + 1)] + x0[IX(i, j - 1)] - 4 * x0[IX(i, j)]);
        }
    }
    set_bnd(N, b, x);
}

void advect(int N, int b, float *d, float *d0, float *u, float *v, float dt)
{
    int i, j, i0, j0, i1, j1;
    float x, y, s0, t0, s1, t1, dt0;
    dt0 = dt * N;
    for (i = 1; i <= N; i++)
    {
        for (j = 1; j <= N; j++)
        {
            x = i - dt0 * u[IX(i, j)];
            y = j - dt0 * v[IX(i, j)];
            if (x < 0.5)
                x = 0.5;
            if (x > N + 0.5)
                x = N + 0.5;
            i0 = (int)x;
            i1 = i0 + 1;
            if (y < 0.5)
                y = 0.5;
            if (y > N + 0.5)
                y = N + 0.5;
            j0 = (int)y;
            j1 = j0 + 1;
            s1 = x - i0;
            s0 = 1 - s1;
            t1 = y - j0;
            t0 = 1 - t1;
            d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
                          s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
        }
    }
    set_bnd(N, b, d);
}

void dens_step(int N, float *x, float *x0, float *u, float *v, float diff,
               float dt)
{
    add_source(N, x, x0, dt);
    SWAP(x0, x);
    diffuse(N, 0, x, x0, diff, dt);
    SWAP(x0, x);
    advect(N, 0, x, x0, u, v, dt);
}
// num of threads = 256
void vel_step(int N, float *u, float *v, float *u0, float *v0,
              float visc, float dt, float *p_new)
{
    add_source(N, u, u0, dt);
    add_source(N, v, v0, dt);
    SWAP(u0, u);
    diffuse(N, 1, u, u0, visc, dt);
    SWAP(v0, v);
    diffuse(N, 2, v, v0, visc, dt);
    // dim3 gridDim(N/16, N/16)
    // dim3 blockDim(16, 16)
    // row = blockIdx.y * blockDim.y + threadIdx.y;
    // col = blockIdx.x * blockDim.x + threadIdx.x;
    // u[row][col]
    project(N, u, v, u0, v0, p_new);
    SWAP(u0, u);
    SWAP(v0, v);
    advect(N, 1, u, u0, u0, v0, dt);
    advect(N, 2, v, v0, u0, v0, dt);
    project(N, u, v, u0, v0, p_new);
}

int main()
{
    auto start_time = std::chrono::steady_clock::now();
    int simulating = 1;
    const int N = 100;
    const int size = (N + 2) * (N + 2);
    float static u[size], v[size];
    float static u_prev[size]; // = {[0 ... 15] = 1000.0};
    float static v_prev[size]; // = {[0 ... 15] = 1000.0};
    float static dens[size], dens_prev[size];
    float static p_new[size];

    std::fill(u_prev, u_prev + size, 100.0);
    std::fill(v_prev, v_prev + size, 100.0);
    std::fill(dens_prev, dens_prev + size, 100.0);
    float dt = 0.01;
    float visc = 0.00001;
    float diff = 0.00001;
    while (simulating--)
    {
        // get_from_UI(dens_prev, u_prev, v_prev);
        vel_step(N, u, v, u_prev, v_prev, visc, dt, p_new);
        dens_step(N, dens, dens_prev, u, v, diff, dt);
        using namespace std;
        //cout << u[5] << endl;
        //  draw_dens(N, dens);
    }
    auto end_time = std::chrono::steady_clock::now();
    std::chrono::duration<double> difference = end_time - start_time;
    double seconds = difference.count();
    std::cout << "Simulation Time = " << seconds << " seconds for " << N << " blocks.\n";
}