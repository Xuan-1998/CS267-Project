#include <hip/hip_runtime.h> 

#include <iostream>
#include <algorithm>

using floating_t = double;
#define IX(i, j) ((i) + (N + 2) * (j))
#define SWAP(x0, x)      \
    {                    \
        floating_t *tmp = x0; \
        x0 = x;          \
        x = tmp;         \
    }


__device__ floating_t testvar = 5;

/*
const int size = N;
*/
void project(int N, floating_t *u, floating_t *v, floating_t *p, floating_t *div);

void add_source(int N, floating_t *x, floating_t *s, floating_t dt)
{
    int i, size = (N + 2) * (N + 2);
    for (i = 0; i < size; i++)
        x[i] += dt * s[i];
}

void set_bnd(int N, int b, floating_t *x)
{
    int i;
    for (i = 1; i <= N; i++)
    {
        x[IX(0, i)] = b == 1 ? -x[IX(1, i)] : x[IX(1, i)];
        x[IX(N + 1, i)] = b == 1 ? -x[IX(N, i)] : x[IX(N, i)];
        x[IX(i, 0)] = b == 2 ? -x[IX(i, 1)] : x[IX(i, 1)];
        x[IX(i, N + 1)] = b == 2 ? -x[IX(i, N)] : x[IX(i, N)];
    }
    x[IX(0, 0)] = 0.5 * (x[IX(1, 0)] + x[IX(0, 1)]);
    x[IX(0, N + 1)] = 0.5 * (x[IX(1, N + 1)] + x[IX(0, N)]);
    x[IX(N + 1, 0)] = 0.5 * (x[IX(N, 0)] + x[IX(N + 1, 1)]);
    x[IX(N + 1, N + 1)] = 0.5 * (x[IX(N, N + 1)] + x[IX(N + 1, N)]);
}

void diffuse(int N, int b, floating_t *x, floating_t *x0, floating_t diff, floating_t dt)
{
    int i, j, k;
    floating_t a = dt * diff * N * N;
    for (k = 0; k < 20; k++)
    {
        for (i = 1; i <= N; i++)
        {
            for (j = 1; j <= N; j++)
            {
                x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] +
                                                   x[IX(i, j - 1)] + x[IX(i, j + 1)])) /
                              (1 + 4 * a);
            }
        }
        set_bnd(N, b, x);
    }
}

void advect(int N, int b, floating_t *d, floating_t *d0, floating_t *u, floating_t *v, floating_t dt)
{
    int i, j, i0, j0, i1, j1;
    floating_t x, y, s0, t0, s1, t1, dt0;
    dt0 = dt * N;
    for (i = 1; i <= N; i++)
    {
        for (j = 1; j <= N; j++)
        {
            x = i - dt0 * u[IX(i, j)];
            y = j - dt0 * v[IX(i, j)];
            if (x < 0.5)
                x = 0.5;
            if (x > N + 0.5)
                x = N + 0.5;
            i0 = (int)x;
            i1 = i0 + 1;
            if (y < 0.5)
                y = 0.5;
            if (y > N + 0.5)
                y = N + 0.5;
            j0 = (int)y;
            j1 = j0 + 1;
            s1 = x - i0;
            s0 = 1 - s1;
            t1 = y - j0;
            t0 = 1 - t1;
            d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
                          s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
        }
    }
    set_bnd(N, b, d);
}

void dens_step(int N, floating_t *x, floating_t *x0, floating_t *u, floating_t *v, floating_t diff,
               floating_t dt)
{
    add_source(N, x, x0, dt);
    SWAP(x0, x);
    diffuse(N, 0, x, x0, diff, dt);
    SWAP(x0, x);
    advect(N, 0, x, x0, u, v, dt);
}

void vel_step(int N, floating_t *u, floating_t *v, floating_t *u0, floating_t *v0,
              floating_t visc, floating_t dt)
{
    add_source(N, u, u0, dt);
    add_source(N, v, v0, dt);
    SWAP(u0, u);
    diffuse(N, 1, u, u0, visc, dt);
    SWAP(v0, v);
    diffuse(N, 2, v, v0, visc, dt);
    project(N, u, v, u0, v0);
    SWAP(u0, u);
    SWAP(v0, v);
    advect(N, 1, u, u0, u0, v0, dt);
    advect(N, 2, v, v0, u0, v0, dt);
    project(N, u, v, u0, v0);
}

void project(int N, floating_t *u, floating_t *v, floating_t *p, floating_t *div)
{
    int i, j, k;
    floating_t h;
    h = 1.0 / N;
    for (i = 1; i <= N; i++)
    {
        for (j = 1; j <= N; j++)
        {
            div[IX(i, j)] = -0.5 * h * (u[IX(i + 1, j)] - u[IX(i - 1, j)] + v[IX(i, j + 1)] - v[IX(i, j - 1)]);
            p[IX(i, j)] = 0;
        }
    }
    set_bnd(N, 0, div);
    set_bnd(N, 0, p);
    for (k = 0; k < 20; k++)
    {
        for (i = 1; i <= N; i++)
        {
            for (j = 1; j <= N; j++)
            {
                p[IX(i, j)] = (div[IX(i, j)] + p[IX(i - 1, j)] + p[IX(i + 1, j)] +
                               p[IX(i, j - 1)] + p[IX(i, j + 1)]) /
                              4;
            }
        }
        set_bnd(N, 0, p);
    }
    for (i = 1; i <= N; i++)
    {
        for (j = 1; j <= N; j++)
        {
            u[IX(i, j)] -= 0.5 * (p[IX(i + 1, j)] - p[IX(i - 1, j)]) / h;
            v[IX(i, j)] -= 0.5 * (p[IX(i, j + 1)] - p[IX(i, j - 1)]) / h;
        }
    }
    set_bnd(N, 1, u);
    set_bnd(N, 2, v);
}

int main()
{
    int simulating = 1000;
    const int N = 1000;
    const int size = (N + 2) * (N + 2);
    floating_t static u[size]{}, v[size]{};
    floating_t static u_prev[size]{}; // = {[0 ... 15] = 1000.0};
    floating_t static v_prev[size]{}; // = {[0 ... 15] = 1000.0};
    floating_t static dens[size]{}, dens_prev[size]{};

    std::fill(u_prev, u_prev + size, 100.0);
    std::fill(v_prev, v_prev + size, 100.0);
    std::fill(dens_prev, dens_prev + size, 100.0);
    constexpr floating_t dt = 0.01;
    constexpr floating_t visc = 0.1;
    constexpr floating_t diff = 1;
    while (simulating--)
    {
        // get_from_UI(dens_prev, u_prev, v_prev);
        vel_step(N, u, v, u_prev, v_prev, visc, dt);
        dens_step(N, dens, dens_prev, u, v, diff, dt);
        using namespace std;
        cout << u[5] << endl;
        // draw_dens(N, dens);
    }
}